#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>


THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_BEGIN


template <typename Iterator1, typename Iterator2>
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA || THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_HIP
__global__
THRUST_HIP_LAUNCH_BOUNDS_DEFAULT
#endif
void simple_copy_on_device(Iterator1 first1, Iterator1 last1, Iterator2 first2)
{
    while(first1 != last1)
        *(first2++) = *(first1++);
}

template <typename Iterator1, typename Iterator2>
void simple_copy(Iterator1 first1, Iterator1 last1, Iterator2 first2)
{
#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA || THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_HIP
    simple_copy_on_device<<<1,1>>>(first1, last1, first2);
#else
    simple_copy_on_device(first1, last1, first2);
#endif
}


void TestDeviceDereferenceDeviceVectorIterator(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100);
    thrust::device_vector<int> output(input.size(), 0);

    simple_copy(input.begin(), input.end(), output.begin());

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceDeviceVectorIterator);

void TestDeviceDereferenceDevicePtr(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100);
    thrust::device_vector<int> output(input.size(), 0);

    thrust::device_ptr<int> _first1 = &input[0];
    thrust::device_ptr<int> _last1  = _first1 + input.size();
    thrust::device_ptr<int> _first2 = &output[0];

    simple_copy(_first1, _last1, _first2);

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceDevicePtr);

void TestDeviceDereferenceTransformIterator(void)
{
    thrust::device_vector<int> input = unittest::random_integers<int>(100);
    thrust::device_vector<int> output(input.size(), 0);

    simple_copy(thrust::make_transform_iterator(input.begin(), thrust::identity<int>()),
                thrust::make_transform_iterator(input.end (),  thrust::identity<int>()),
                output.begin());

    ASSERT_EQUAL(input, output);
}
DECLARE_UNITTEST(TestDeviceDereferenceTransformIterator);

void TestDeviceDereferenceCountingIterator(void)
{
    thrust::counting_iterator<int> first(1);
    thrust::counting_iterator<int> last(6);

    thrust::device_vector<int> output(5);

    simple_copy(first, last, output.begin());

    ASSERT_EQUAL(output[0], 1);
    ASSERT_EQUAL(output[1], 2);
    ASSERT_EQUAL(output[2], 3);
    ASSERT_EQUAL(output[3], 4);
    ASSERT_EQUAL(output[4], 5);
}
DECLARE_UNITTEST(TestDeviceDereferenceCountingIterator);

void TestDeviceDereferenceTransformedCountingIterator(void)
{
    thrust::counting_iterator<int> first(1);
    thrust::counting_iterator<int> last(6);

    thrust::device_vector<int> output(5);

    simple_copy(thrust::make_transform_iterator(first, thrust::negate<int>()),
                thrust::make_transform_iterator(last,  thrust::negate<int>()),
                output.begin());

    ASSERT_EQUAL(output[0], -1);
    ASSERT_EQUAL(output[1], -2);
    ASSERT_EQUAL(output[2], -3);
    ASSERT_EQUAL(output[3], -4);
    ASSERT_EQUAL(output[4], -5);
}
DECLARE_UNITTEST(TestDeviceDereferenceTransformedCountingIterator);

THRUST_DISABLE_MSVC_POSSIBLE_LOSS_OF_DATA_WARNING_END
