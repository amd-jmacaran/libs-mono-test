/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/runtime_static_assert.h>
#include <unittest/unittest.h>
#include <thrust/generate.h>

struct generator
{
    THRUST_HOST_DEVICE
    int operator()() const
    {
        return 1;
    }
};

void TestGenerateConstIteratorCompilationError()
{
    thrust::host_vector<int> test1(10);

    ASSERT_STATIC_ASSERT(thrust::generate(test1.cbegin(), test1.cend(), generator()));
    ASSERT_STATIC_ASSERT(thrust::generate_n(test1.cbegin(), 10, generator()));
}
DECLARE_UNITTEST(TestGenerateConstIteratorCompilationError);

void TestFillConstIteratorCompilationError()
{
    thrust::host_vector<int> test1(10);
    ASSERT_STATIC_ASSERT(thrust::fill(test1.cbegin(), test1.cend(), 1));
}
DECLARE_UNITTEST(TestFillConstIteratorCompilationError);

