/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <thrust/detail/config.h>

#if THRUST_CPP_DIALECT >= 2014

#include <async/test_policy_overloads.h>

#include <async/exclusive_scan/mixin.h>

// Compilation test with discard iterators. No runtime validation is actually
// performed, other than testing whether the algorithm completes without
// exception.

template <typename input_value_type,
          typename initial_value_type  = input_value_type,
          typename alternate_binary_op = thrust::maximum<>>
struct discard_invoker
    : testing::async::mixin::input::device_vector<input_value_type>
    , testing::async::mixin::output::discard_iterator
    , testing::async::exclusive_scan::mixin::postfix_args::
        all_overloads<initial_value_type, alternate_binary_op>
    , testing::async::mixin::invoke_reference::noop
    , testing::async::exclusive_scan::mixin::invoke_async::simple
    , testing::async::mixin::compare_outputs::noop
{
  static std::string description() { return "discard output"; }
};

template <typename T>
struct test_discard
{
  void operator()(std::size_t num_values) const
  {
    testing::async::test_policy_overloads<discard_invoker<T>>::run(num_values);
  }
};
DECLARE_GENERIC_SIZED_UNITTEST_WITH_TYPES(test_discard, NumericTypes);

#endif // C++14
