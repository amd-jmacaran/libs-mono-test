/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>

#include <thrust/sequence.h>
#include <thrust/sort.h>

//////////////////////
// Scalar Functions //
//////////////////////

template <class Vector>
void TestScalarLowerBoundDescendingSimple(void)
{
    using T = typename Vector::value_type;

    Vector vec(5);

    vec[0] = 8;
    vec[1] = 7;
    vec[2] = 5;
    vec[3] = 2;
    vec[4] = 0;

    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::lower_bound(vec.begin(), vec.end(), T{0}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::lower_bound(vec.begin(), vec.end(), T{1}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::lower_bound(vec.begin(), vec.end(), T{2}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::lower_bound(vec.begin(), vec.end(), T{3}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::lower_bound(vec.begin(), vec.end(), T{4}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::lower_bound(vec.begin(), vec.end(), T{5}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::lower_bound(vec.begin(), vec.end(), T{6}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 1, thrust::lower_bound(vec.begin(), vec.end(), T{7}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::lower_bound(vec.begin(), vec.end(), T{8}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::lower_bound(vec.begin(), vec.end(), T{9}, thrust::greater<T>()));
}
DECLARE_VECTOR_UNITTEST(TestScalarLowerBoundDescendingSimple);


template <class Vector>
void TestScalarUpperBoundDescendingSimple(void)
{
    using T = typename Vector::value_type;

    Vector vec(5);

    vec[0] = 8;
    vec[1] = 7;
    vec[2] = 5;
    vec[3] = 2;
    vec[4] = 0;

    ASSERT_EQUAL_QUIET(vec.begin() + 5, thrust::upper_bound(vec.begin(), vec.end(), T{0}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::upper_bound(vec.begin(), vec.end(), T{1}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::upper_bound(vec.begin(), vec.end(), T{2}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::upper_bound(vec.begin(), vec.end(), T{3}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::upper_bound(vec.begin(), vec.end(), T{4}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::upper_bound(vec.begin(), vec.end(), T{5}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::upper_bound(vec.begin(), vec.end(), T{6}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::upper_bound(vec.begin(), vec.end(), T{7}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 1, thrust::upper_bound(vec.begin(), vec.end(), T{8}, thrust::greater<T>()));
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::upper_bound(vec.begin(), vec.end(), T{9}, thrust::greater<T>()));
}
DECLARE_VECTOR_UNITTEST(TestScalarUpperBoundDescendingSimple);


template <class Vector>
void TestScalarBinarySearchDescendingSimple(void)
{
    using T = typename Vector::value_type;

    Vector vec(5);

    vec[0] = 8;
    vec[1] = 7;
    vec[2] = 5;
    vec[3] = 2;
    vec[4] = 0;

    ASSERT_EQUAL(true,  thrust::binary_search(vec.begin(), vec.end(), T{0}, thrust::greater<T>()));
    ASSERT_EQUAL(false, thrust::binary_search(vec.begin(), vec.end(), T{1}, thrust::greater<T>()));
    ASSERT_EQUAL(true,  thrust::binary_search(vec.begin(), vec.end(), T{2}, thrust::greater<T>()));
    ASSERT_EQUAL(false, thrust::binary_search(vec.begin(), vec.end(), T{3}, thrust::greater<T>()));
    ASSERT_EQUAL(false, thrust::binary_search(vec.begin(), vec.end(), T{4}, thrust::greater<T>()));
    ASSERT_EQUAL(true,  thrust::binary_search(vec.begin(), vec.end(), T{5}, thrust::greater<T>()));
    ASSERT_EQUAL(false, thrust::binary_search(vec.begin(), vec.end(), T{6}, thrust::greater<T>()));
    ASSERT_EQUAL(true,  thrust::binary_search(vec.begin(), vec.end(), T{7}, thrust::greater<T>()));
    ASSERT_EQUAL(true,  thrust::binary_search(vec.begin(), vec.end(), T{8}, thrust::greater<T>()));
    ASSERT_EQUAL(false, thrust::binary_search(vec.begin(), vec.end(), T{9}, thrust::greater<T>()));
}
DECLARE_VECTOR_UNITTEST(TestScalarBinarySearchDescendingSimple);


template <class Vector>
void TestScalarEqualRangeDescendingSimple(void)
{
    using T = typename Vector::value_type;

    Vector vec(5);

    vec[0] = 8;
    vec[1] = 7;
    vec[2] = 5;
    vec[3] = 2;
    vec[4] = 0;

    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::equal_range(vec.begin(), vec.end(), T{0}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::equal_range(vec.begin(), vec.end(), T{1}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{2}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{3}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{4}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::equal_range(vec.begin(), vec.end(), T{5}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::equal_range(vec.begin(), vec.end(), T{6}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 1, thrust::equal_range(vec.begin(), vec.end(), T{7}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::equal_range(vec.begin(), vec.end(), T{8}, thrust::greater<T>()).first);
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::equal_range(vec.begin(), vec.end(), T{9}, thrust::greater<T>()).first);

    ASSERT_EQUAL_QUIET(vec.begin() + 5, thrust::equal_range(vec.begin(), vec.end(), T{0}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::equal_range(vec.begin(), vec.end(), T{1}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 4, thrust::equal_range(vec.begin(), vec.end(), T{2}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{3}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{4}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 3, thrust::equal_range(vec.begin(), vec.end(), T{5}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::equal_range(vec.begin(), vec.end(), T{6}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 2, thrust::equal_range(vec.begin(), vec.end(), T{7}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 1, thrust::equal_range(vec.begin(), vec.end(), T{8}, thrust::greater<T>()).second);
    ASSERT_EQUAL_QUIET(vec.begin() + 0, thrust::equal_range(vec.begin(), vec.end(), T{9}, thrust::greater<T>()).second);
}
DECLARE_VECTOR_UNITTEST(TestScalarEqualRangeDescendingSimple);

