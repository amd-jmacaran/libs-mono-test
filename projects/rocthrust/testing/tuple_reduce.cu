/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

using namespace unittest;

struct SumTupleFunctor
{
  template <typename Tuple>
  THRUST_HOST_DEVICE
  Tuple operator()(const Tuple &lhs, const Tuple &rhs)
  {
    using thrust::get;
  
    return thrust::make_tuple(get<0>(lhs) + get<0>(rhs),
                              get<1>(lhs) + get<1>(rhs));
  }
};

struct MakeTupleFunctor
{
  template<typename T1, typename T2>
  THRUST_HOST_DEVICE
  thrust::tuple<T1,T2> operator()(T1 &lhs, T2 &rhs)
  {
    return thrust::make_tuple(lhs, rhs);
  }
};

template <typename T>
struct TestTupleReduce
{
  void operator()(const size_t n)
  {
     using namespace thrust;

     host_vector<T> h_t1 = random_integers<T>(n);
     host_vector<T> h_t2 = random_integers<T>(n);

     // zip up the data
     host_vector< tuple<T,T> > h_tuples(n);
     transform(h_t1.begin(), h_t1.end(), h_t2.begin(), h_tuples.begin(), MakeTupleFunctor());

     // copy to device
     device_vector< tuple<T,T> > d_tuples = h_tuples;
     
     tuple<T,T> zero(0,0);

     // sum on host
     tuple<T,T> h_result = reduce(h_tuples.begin(), h_tuples.end(), zero, SumTupleFunctor());

     // sum on device
     tuple<T,T> d_result = reduce(d_tuples.begin(), d_tuples.end(), zero, SumTupleFunctor());

     ASSERT_EQUAL_QUIET(h_result, d_result);
  }
};
VariableUnitTest<TestTupleReduce, IntegralTypes> TestTupleReduceInstance;

