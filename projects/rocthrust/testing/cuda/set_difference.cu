#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Iterator3, typename Iterator4>
__global__
void set_difference_kernel(ExecutionPolicy exec, Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator2 last2, Iterator3 result1, Iterator4 result2)
{
  *result2 = thrust::set_difference(exec, first1, last1, first2, last2, result1);
}


template<typename ExecutionPolicy>
void TestSetDifferenceDevice(ExecutionPolicy exec)
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = typename Vector::iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  thrust::device_vector<Iterator> end_vec(1);

  set_difference_kernel<<<1,1>>>(exec, a.begin(), a.end(), b.begin(), b.end(), result.begin(), end_vec.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  Iterator end = end_vec.front();

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}


void TestSetDifferenceDeviceSeq()
{
  TestSetDifferenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSetDifferenceDeviceSeq);


void TestSetDifferenceDeviceDevice()
{
  TestSetDifferenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSetDifferenceDeviceDevice);
#endif


void TestSetDifferenceCudaStreams()
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = Vector::iterator;

  Vector a(4), b(5);

  a[0] = 0; a[1] = 2; a[2] = 4; a[3] = 5;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4; b[4] = 6;

  Vector ref(2);
  ref[0] = 2; ref[1] = 5;

  Vector result(2);

  hipStream_t s;
  hipStreamCreate(&s);

  Iterator end = thrust::set_difference(thrust::cuda::par.on(s), a.begin(), a.end(), b.begin(), b.end(), result.begin());
  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSetDifferenceCudaStreams);

