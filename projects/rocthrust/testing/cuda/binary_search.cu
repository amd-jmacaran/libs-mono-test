/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>

void TestEqualRangeOnStream()
{ // Regression test for GH issue #921 (nvbug 2173437)
  using vector_t   = typename thrust::device_vector<int>;
  using iterator_t = typename vector_t::iterator;
  using result_t   = thrust::pair<iterator_t, iterator_t>;

  vector_t input(10);
  thrust::sequence(thrust::device, input.begin(), input.end(), 0);
  hipStream_t stream = 0;
  result_t result = thrust::equal_range(thrust::cuda::par.on(stream),
                                        input.begin(), input.end(),
                                        5);

  ASSERT_EQUAL(5, thrust::distance(input.begin(), result.first));
  ASSERT_EQUAL(6, thrust::distance(input.begin(), result.second));
}
DECLARE_UNITTEST(TestEqualRangeOnStream);
