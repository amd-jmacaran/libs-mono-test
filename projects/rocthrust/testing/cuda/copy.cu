#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::copy(exec, first, last, result);
}


template<typename T, typename ExecutionPolicy>
void TestCopyDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_src = unittest::random_integers<T>(n);
  thrust::host_vector<T>   h_dst(n);

  thrust::device_vector<T> d_src = h_src;
  thrust::device_vector<T> d_dst(n);
  
  thrust::copy(h_src.begin(), h_src.end(), h_dst.begin());
  copy_kernel<<<1,1>>>(exec, d_src.begin(), d_src.end(), d_dst.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_dst, d_dst);
}


template<typename T>
void TestCopyDeviceSeq(size_t n)
{
  TestCopyDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyDeviceSeq);


template<typename T>
void TestCopyDeviceDevice(size_t n)
{
  TestCopyDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyDeviceDevice);


template<typename ExecutionPolicy, typename Iterator1, typename Size, typename Iterator2>
__global__
void copy_n_kernel(ExecutionPolicy exec, Iterator1 first, Size n, Iterator2 result)
{
  thrust::copy_n(exec, first, n, result);
}


template<typename T, typename ExecutionPolicy>
void TestCopyNDevice(ExecutionPolicy exec, size_t n)
{
  thrust::host_vector<T>   h_src = unittest::random_integers<T>(n);
  thrust::host_vector<T>   h_dst(n);

  thrust::device_vector<T> d_src = h_src;
  thrust::device_vector<T> d_dst(n);
  
  thrust::copy_n(h_src.begin(), h_src.size(), h_dst.begin());
  copy_n_kernel<<<1,1>>>(exec, d_src.begin(), d_src.size(), d_dst.begin());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_dst, d_dst);
}


template<typename T>
void TestCopyNDeviceSeq(size_t n)
{
  TestCopyNDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyNDeviceSeq);


template<typename T>
void TestCopyNDeviceDevice(size_t n)
{
  TestCopyNDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestCopyNDeviceDevice);
#endif

