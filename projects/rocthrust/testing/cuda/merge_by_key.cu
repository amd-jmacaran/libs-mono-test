#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy,
         typename Iterator1,
         typename Iterator2,
         typename Iterator3,
         typename Iterator4,
         typename Iterator5,
         typename Iterator6,
         typename Iterator7>
__global__
void merge_by_key_kernel(ExecutionPolicy exec,
                         Iterator1 keys_first1, Iterator1 keys_last1,
                         Iterator2 keys_first2, Iterator2 keys_last2,
                         Iterator3 values_first1,
                         Iterator4 values_first2,
                         Iterator5 keys_result,
                         Iterator6 values_result,
                         Iterator7 result)
{
  *result = thrust::merge_by_key(exec, keys_first1, keys_last1, keys_first2, keys_last2, values_first1, values_first2, keys_result, values_result);
}


template<typename ExecutionPolicy>
void TestMergeByKeyDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> a_key(3), a_val(3), b_key(4), b_val(4);

  a_key[0] = 0;  a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 13; a_val[1] = 7; a_val[2] = 42;

  b_key[0] = 0 ; b_key[1] = 3;  b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 42; b_val[1] = 42; b_val[2] = 7; b_val[3] = 13;

  thrust::device_vector<int> ref_key(7), ref_val(7);
  ref_key[0] = 0; ref_val[0] = 13;
  ref_key[1] = 0; ref_val[1] = 42;
  ref_key[2] = 2; ref_val[2] = 7;
  ref_key[3] = 3; ref_val[3] = 42;
  ref_key[4] = 3; ref_val[4] = 7;
  ref_key[5] = 4; ref_val[5] = 42;
  ref_key[6] = 4; ref_val[6] = 13;

  thrust::device_vector<int> result_key(7), result_val(7);

  using Iterator = typename thrust::device_vector<int>::iterator;

  thrust::device_vector<thrust::pair<Iterator,Iterator> > result_ends(1);

  merge_by_key_kernel<<<1,1>>>(exec,
                               a_key.begin(), a_key.end(),
                               b_key.begin(), b_key.end(),
                               a_val.begin(), b_val.begin(),
                               result_key.begin(),
                               result_val.begin(),
                               result_ends.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  thrust::pair<Iterator,Iterator> ends = result_ends[0];

  ASSERT_EQUAL_QUIET(result_key.end(), ends.first);
  ASSERT_EQUAL_QUIET(result_val.end(), ends.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);
}


void TestMergeByKeyDeviceSeq()
{
  TestMergeByKeyDevice(thrust::seq);
}
DECLARE_UNITTEST(TestMergeByKeyDeviceSeq);


void TestMergeByKeyDeviceDevice()
{
  TestMergeByKeyDevice(thrust::device);
}
DECLARE_UNITTEST(TestMergeByKeyDeviceDevice);
#endif


void TestMergeByKeyCudaStreams()
{
  using Vector   = thrust::device_vector<int>;
  using Iterator = Vector::iterator;

  Vector a_key(3), a_val(3), b_key(4), b_val(4);

  a_key[0] = 0;  a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 13; a_val[1] = 7; a_val[2] = 42;

  b_key[0] = 0 ; b_key[1] = 3;  b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 42; b_val[1] = 42; b_val[2] = 7; b_val[3] = 13;

  Vector ref_key(7), ref_val(7);
  ref_key[0] = 0; ref_val[0] = 13;
  ref_key[1] = 0; ref_val[1] = 42;
  ref_key[2] = 2; ref_val[2] = 7;
  ref_key[3] = 3; ref_val[3] = 42;
  ref_key[4] = 3; ref_val[4] = 7;
  ref_key[5] = 4; ref_val[5] = 42;
  ref_key[6] = 4; ref_val[6] = 13;

  Vector result_key(7), result_val(7);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::pair<Iterator,Iterator> ends =
    thrust::merge_by_key(thrust::cuda::par.on(s),
                         a_key.begin(), a_key.end(),
                         b_key.begin(), b_key.end(),
                         a_val.begin(), b_val.begin(),
                         result_key.begin(),
                         result_val.begin());

  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result_key.end(), ends.first);
  ASSERT_EQUAL_QUIET(result_val.end(), ends.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMergeByKeyCudaStreams);

