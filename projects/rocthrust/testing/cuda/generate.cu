#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/generate.h>
#include <thrust/execution_policy.h>


template<typename T>
struct return_value
{
  T val;

  return_value() {}
  return_value(T v)
      : val(v)
  {}

  THRUST_HOST_DEVICE T operator()(void)
  {
    return val;
  }
};


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Function>
__global__
void generate_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f)
{
  thrust::generate(exec, first, last, f);
}


template<typename T, typename ExecutionPolicy>
void TestGenerateDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);
  
  T value = 13;
  return_value<T> f(value);
  
  thrust::generate(h_result.begin(), h_result.end(), f);

  generate_kernel<<<1,1>>>(exec, d_result.begin(), d_result.end(), f);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_result, d_result);
}


template<typename T>
void TestGenerateDeviceSeq(const size_t n)
{
  TestGenerateDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateDeviceSeq);


template<typename T>
void TestGenerateDeviceDevice(const size_t n)
{
  TestGenerateDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateDeviceDevice);
#endif


void TestGenerateCudaStreams()
{
  thrust::device_vector<int> result(5);
  
  int value = 13;
  
  return_value<int> f(value);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::generate(thrust::cuda::par.on(s), result.begin(), result.end(), f);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(result[0], value);
  ASSERT_EQUAL(result[1], value);
  ASSERT_EQUAL(result[2], value);
  ASSERT_EQUAL(result[3], value);
  ASSERT_EQUAL(result[4], value);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGenerateCudaStreams);


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator, typename Size, typename Function>
__global__
void generate_n_kernel(ExecutionPolicy exec, Iterator first, Size n, Function f)
{
  thrust::generate_n(exec, first, n, f);
}


template<typename T, typename ExecutionPolicy>
void TestGenerateNDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);
  
  T value = 13;
  return_value<T> f(value);
  
  thrust::generate_n(h_result.begin(), h_result.size(), f);

  generate_n_kernel<<<1,1>>>(exec, d_result.begin(), d_result.size(), f);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(h_result, d_result);
}


template<typename T>
void TestGenerateNDeviceSeq(const size_t n)
{
  TestGenerateNDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateNDeviceSeq);


template<typename T>
void TestGenerateNDeviceDevice(const size_t n)
{
  TestGenerateNDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestGenerateNDeviceDevice);
#endif


void TestGenerateNCudaStreams()
{
  thrust::device_vector<int> result(5);
  
  int value = 13;
  
  return_value<int> f(value);

  hipStream_t s;
  hipStreamCreate(&s);
  
  thrust::generate_n(thrust::cuda::par.on(s), result.begin(), result.size(), f);
  hipStreamSynchronize(s);
  
  ASSERT_EQUAL(result[0], value);
  ASSERT_EQUAL(result[1], value);
  ASSERT_EQUAL(result[2], value);
  ASSERT_EQUAL(result[3], value);
  ASSERT_EQUAL(result[4], value);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestGenerateNCudaStreams);

