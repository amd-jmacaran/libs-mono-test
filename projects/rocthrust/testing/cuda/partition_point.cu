#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/partition.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator1, typename Predicate, typename Iterator2>
__global__
void partition_point_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Predicate pred, Iterator2 result)
{
  *result = thrust::partition_point(exec, first, last, pred);
}


template<typename T>
struct is_even
{
  THRUST_HOST_DEVICE bool operator()(T x) const
  {
    return ((int) x % 2) == 0;
  }
};


template<typename ExecutionPolicy>
void TestPartitionPointDevice(ExecutionPolicy exec)
{
  size_t n = 1000;
  thrust::device_vector<int> v = unittest::random_integers<int>(n);
  using iterator               = typename thrust::device_vector<int>::iterator;

  iterator ref = thrust::stable_partition(v.begin(), v.end(), is_even<int>());

  thrust::device_vector<iterator> result(1);
  partition_point_kernel<<<1,1>>>(exec, v.begin(), v.end(), is_even<int>(), result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);

  ASSERT_EQUAL(ref - v.begin(), (iterator)result[0] - v.begin());
}


void TestPartitionPointDeviceSeq()
{
  TestPartitionPointDevice(thrust::seq);
}
DECLARE_UNITTEST(TestPartitionPointDeviceSeq);


void TestPartitionPointDeviceDevice()
{
  TestPartitionPointDevice(thrust::device);
}
DECLARE_UNITTEST(TestPartitionPointDeviceDevice);
#endif


void TestPartitionPointCudaStreams()
{
  using Vector   = thrust::device_vector<int>;
  using T        = Vector::value_type;
  using Iterator = Vector::iterator;

  Vector v(4);
  v[0] = 1; v[1] = 1; v[2] = 1; v[3] = 0;

  Iterator first = v.begin();

  Iterator last = v.begin() + 4;
  Iterator ref = first + 3;

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::cuda::par.on(s), first, last, thrust::identity<T>()));

  last = v.begin() + 3;
  ref = last;
  ASSERT_EQUAL_QUIET(ref, thrust::partition_point(thrust::cuda::par.on(s), first, last, thrust::identity<T>()));

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestPartitionPointCudaStreams);

