#include "hip/hip_runtime.h"
/*
 *  Copyright 2008-2013 NVIDIA Corporation
 *  Modifications Copyright© 2019-2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <unittest/unittest.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>


#ifdef THRUST_TEST_DEVICE_SIDE
template<typename ExecutionPolicy, typename Iterator>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::sequence(exec, first, last);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init)
{
  thrust::sequence(exec, first, last, init);
}


template<typename ExecutionPolicy, typename Iterator, typename T>
__global__
void sequence_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, T step)
{
  thrust::sequence(exec, first, last, init, step);
}


template<typename ExecutionPolicy>
void TestSequenceDevice(ExecutionPolicy exec)
{
  thrust::device_vector<int> v(5);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end());
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
 
  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  sequence_kernel<<<1,1>>>(exec, v.begin(), v.end(), 10, 2);
  {
    hipError_t const err = hipDeviceSynchronize();
    ASSERT_EQUAL(hipSuccess, err);
  }
  
  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);
}

void TestSequenceDeviceSeq()
{
  TestSequenceDevice(thrust::seq);
}
DECLARE_UNITTEST(TestSequenceDeviceSeq);

void TestSequenceDeviceDevice()
{
  TestSequenceDevice(thrust::device);
}
DECLARE_UNITTEST(TestSequenceDeviceDevice);
#endif

void TestSequenceCudaStreams()
{
  using Vector = thrust::device_vector<int>;

  Vector v(5);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end());
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 0);
  ASSERT_EQUAL(v[1], 1);
  ASSERT_EQUAL(v[2], 2);
  ASSERT_EQUAL(v[3], 3);
  ASSERT_EQUAL(v[4], 4);

  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 11);
  ASSERT_EQUAL(v[2], 12);
  ASSERT_EQUAL(v[3], 13);
  ASSERT_EQUAL(v[4], 14);
  
  thrust::sequence(thrust::cuda::par.on(s), v.begin(), v.end(), 10, 2);
  hipStreamSynchronize(s);

  ASSERT_EQUAL(v[0], 10);
  ASSERT_EQUAL(v[1], 12);
  ASSERT_EQUAL(v[2], 14);
  ASSERT_EQUAL(v[3], 16);
  ASSERT_EQUAL(v[4], 18);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestSequenceCudaStreams);

